#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "kernel_C.h"

/*
 * This function is used to make sure cuda is ready 
 * before the real execution of the program.
 */
void initializeCuda()
{
  int *d_k;

  //allocate device memory for source array  
  if( hipSuccess != hipMalloc((void **)&d_k,sizeof(int) )) {
    printf("ERROR DURING ALLOCATING MEMORY for k\n");
    exit(1);
  }     
  hipFree(d_k);
  printf("CUDA is initialized\n");
}

