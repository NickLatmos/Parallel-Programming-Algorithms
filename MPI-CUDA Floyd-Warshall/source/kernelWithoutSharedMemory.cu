#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "header.h"
#include "kernel_C.h"

extern int world_size;  //number of processes

__global__ void kernelWithoutSharedMemory(float *A, float *row_k, int *n, int *k);

__global__ void kernelWithoutSharedMemory(float *A, float *row_k, int *n, int *k) 
{ 
  int x = threadIdx.x + blockIdx.x * blockDim.x;   
  int y = threadIdx.y + blockIdx.y * blockDim.y;   
  int id = x + y * gridDim.x * blockDim.x;                      
  int id_i_k = (*k) + y * gridDim.x * blockDim.x;   //id for A(i,k)                                              
  
  if( A[id] > A[id_i_k] + row_k[x] )         
    A[id] = A[id_i_k] + row_k[x];       
}

/*
 * This function calls the device in order to execute the Warshall-Floyd Algorithm
 * It doesn't use shared memory
 */
void executeKernelWithoutSharedMemory(int n, int number_of_rows , int world_rank, float *testArray)
{
  int k, root, *d_n, *d_k, step = 0;
  float *d_A, *row_k, *d_row_k;
  hipError_t hipError_t;

  //128 threads per block => n/128 blocks in y-axis and number_of_rows = n/world_size blocks in x-axis => n/128 * n/world_size * 128 = n^2/world_size threads per process => OK! 
  dim3 dimBlock( 128 , 1 );               //128,1
  dim3 dimGrid( n/dimBlock.x , number_of_rows ); // n/128, number_of_rows

  //allocate device memory for source array  
  if( hipSuccess != hipMalloc((void **)&d_A, n*n/world_size*sizeof(float) )) {
    printf("ERROR DURING ALLOCATING MEMORY for d_A\n");
    exit(1);
  }     
  
  //allocate device memory for source array  
  if( hipSuccess != hipMalloc((void **)&d_row_k, n*sizeof(float)) ){
    printf("ERROR DURING ALLOCATING MEMORY for d_row\n");
    exit(1);
  }

  //allocate device memory
  if( hipSuccess != hipMalloc((void **)&d_n, sizeof(int)) ){
    printf("ERROR DURING ALLOCATING MEMORY for d_n\n");
    exit(1);
  }

  //allocate device memory   
  if( hipSuccess != hipMalloc((void **)&d_k, sizeof(int)) ){
    printf("ERROR DURING ALLOCATING MEMORY for d_k\n");
    exit(1);
  }

  // Copy inputs to the device 
  hipMemcpy(d_A, testArray, n*n/world_size*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_n, &n, sizeof(int), hipMemcpyHostToDevice);

  row_k = (float *)malloc(n*sizeof(float));

  //call kernel
  for(k = 0; k < n; k++){

    hipMemcpy(d_k, &k, sizeof(int), hipMemcpyHostToDevice);

    //The process that own's the k'th row must broadcast this row to everyone else
    root = k/number_of_rows;

    //Save the row in an array. Only for root
    if(world_rank == root){                                           
      for(int j = 0; j < n; j++)                                    
        row_k[j] = testArray[step * n + j]; //linearArray[k*n + j] = A(k,j) which is the entry I must send                                                                
      step++;
    }

    //Receive the row 
    receiveRow(row_k, n, root);
    hipMemcpy(d_row_k, row_k, n*sizeof(float), hipMemcpyHostToDevice); //We must save it in the device memory as it will be used there

    kernelWithoutSharedMemory<<<dimGrid,dimBlock>>>(d_A, d_row_k, d_n, d_k);
    //hipDeviceSynchronize();

    //Copy result back to host 
    hipMemcpy(testArray, d_A, n*n/world_size*sizeof(float), hipMemcpyDeviceToHost);

    /*if(k != n-1)
      hipMemcpy(d_A, testArray, n*n/world_size*sizeof(float), hipMemcpyHostToDevice);*/
  }

  hipError_t = hipGetLastError();
  if (hipSuccess != hipError_t){
    printf("  hipGetLastError() returned %d: %s\n", hipError_t, hipGetErrorString(hipError_t));
    exit(1);
  }

  //free the memory
  hipFree(d_A);
  hipFree(d_row_k);
  hipFree(d_k);
  hipFree(d_n);
}
